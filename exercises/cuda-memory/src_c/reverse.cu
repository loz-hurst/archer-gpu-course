/*
 * This is an introductory exercise in using constant memory
 * and then shared memory. The reserve array idea is from an
*  NVIDIA exercise of the same type.
 *
 * Training material developed by Kevin Stratford
 * Copyright EPCC, The University of Edinburgh, 2017 
 */


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

/* Forward Declaration*/
/* Utility function to check for and report CUDA errors */

void checkCUDAError(const char*);

/*
 * The number of CUDA threads per block to use.
 */

#define THREADS_PER_BLOCK 128

/* The number of integer elements in the array */

static __constant__ int array_size;

/* Reverse the elements in the input array d_in.
 * The total number of threads should be size. */

__global__ void reverseArray(int * d_in, int * d_out)
{

  __shared__ int tmp[THREADS_PER_BLOCK];
  int idx = blockIdx.x*blockDim.x + threadIdx.x;

  tmp[THREADS_PER_BLOCK - (threadIdx.x + 1)] = d_in[idx];

  __syncthreads();

  d_out[array_size - (blockIdx.x+1)*blockDim.x + threadIdx.x] = tmp[threadIdx.x];
}


/* Main routine */
int main(int argc, char *argv[])
{
    int *h_in, *h_out;
    int *d_in, *d_out;

    int i;
    int ncorrect;
    int size = 65536;
    size_t sz = size * sizeof(int);

    /* Print device details */
    int deviceNum;
    hipGetDevice(&deviceNum);
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, deviceNum);
    printf("  Device name: %s\n", prop.name);


    /*
     * allocate memory on host
     * h_in holds the input array, h_out holds the result
     */
    h_in = (int *) malloc(sz);
    h_out = (int *) malloc(sz);

    /*
     * allocate memory on device
     */
    hipMalloc(&d_in, sz);
    hipMalloc(&d_out, sz);

    /* initialise host arrays */
    for (i = 0; i < size; i++) {
        h_in[i] = i;
        h_out[i] = 0;
    }

    // Set constant
    hipMemcpyToSymbol(HIP_SYMBOL(array_size), &size, sizeof(int));

    /* copy input array from host to GPU */

    hipMemcpy(d_in, h_in, sz, hipMemcpyHostToDevice);

    /* run the kernel on the GPU */

    dim3 blocksPerGrid(size/THREADS_PER_BLOCK, 1, 1);
    dim3 threadsPerBlock(THREADS_PER_BLOCK, 1, 1);

    reverseArray<<< blocksPerGrid, threadsPerBlock >>>(d_in, d_out);

    /* wait for all threads to complete and check for errors */

    hipDeviceSynchronize();
    checkCUDAError("kernel invocation");

    /* copy the result array back to the host */

    hipMemcpy(h_out, d_out, sz, hipMemcpyDeviceToHost);
    checkCUDAError("hipMemcpyDeviceToHost");

    /* print out the result */
    printf("Results: ");
    ncorrect = 0;
    for (i = 0; i < size; i++) {
      if (h_out[i] == h_in[size - (i+1)]) ncorrect += 1;
    }
    printf("Number of correctly reversed elements %d (%s)\n", ncorrect,
           ncorrect == size ? "Correct" : "INCORRECT");
    printf("\n");

    /* free device buffers */

    hipFree(d_out);
    hipFree(d_in);

    /* free host buffers */
    free(h_in);
    free(h_out);

    return 0;
}


/* Utility function to check for and report CUDA errors */
void checkCUDAError(const char *msg)
{
    hipError_t err = hipGetLastError();
    if( hipSuccess != err) 
    {
        fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString( err) );
        exit(EXIT_FAILURE);
    }                         
}
